#include "hip/hip_runtime.h"
#include <cstdio>
#include "vector3.h"
#define CUDA_TITAN
#include "hermite6.h"
// #include "hermite6-titan.h"

__device__ __forceinline__ void predict_one(
		const double             tsys,
		const Gravity::GParticle &p,
		Gravity::GPredictor      &pr)
{
		const double dt  = tsys - p.tlast;
		const double dt2 = (1./2.) * dt;
		const double dt3 = (1./3.) * dt;
		const double dt4 = (1./4.) * dt;
		const double dt5 = (1./5.) * dt;

		double3 pos, vel, acc;

#if 1
		pos.x = 
			p.pos.x + dt *(
			p.vel.x + dt2*(
			p.acc.x + dt3*(
			p.jrk.x + dt4*(
			p.snp.x + dt5*(
			p.crk.x )))));
		pos.y = 
			p.pos.y + dt *(
			p.vel.y + dt2*(
			p.acc.y + dt3*(
			p.jrk.y + dt4*(
			p.snp.y + dt5*(
			p.crk.y )))));
		pos.z = 
			p.pos.z + dt *(
			p.vel.z + dt2*(
			p.acc.z + dt3*(
			p.jrk.z + dt4*(
			p.snp.z + dt5*(
			p.crk.z )))));
#endif

#if 1
		vel.x = 
			p.vel.x + dt *(
			p.acc.x + dt2*(
			p.jrk.x + dt3*(
			p.snp.x + dt4*(
			p.crk.x ))));
		vel.y = 
			p.vel.y + dt *(
			p.acc.y + dt2*(
			p.jrk.y + dt3*(
			p.snp.y + dt4*(
			p.crk.y ))));
		vel.z = 
			p.vel.z + dt *(
			p.acc.z + dt2*(
			p.jrk.z + dt3*(
			p.snp.z + dt4*(
			p.crk.z ))));
#endif

#if 1
		acc.x = 
			p.acc.x + dt *(
			p.jrk.x + dt2*(
			p.snp.x + dt3*(
			p.crk.x )));
		acc.y = 
			p.acc.y + dt *(
			p.jrk.y + dt2*(
			p.snp.y + dt3*(
			p.crk.y )));
		acc.z = 
			p.acc.z + dt *(
			p.jrk.z + dt2*(
			p.snp.z + dt3*(
			p.crk.z )));
#endif

		pr.pos  = pos;
		pr.mass = p.mass;
		pr.vel  = vel;
		pr.acc  = acc;
}

#if 0 // naive version
__global__ void predict_kernel(
		const int                 nbody,
		const Gravity::GParticle *ptcl,
		Gravity::GPredictor      *pred,
		const double              tsys)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < nbody){
		Gravity::GParticle   p  = ptcl[tid];
		Gravity::GPredictor &pr = pred[tid];
		predict_one(tsys, p, pr);

	}
}
#else // specialized for 32 threads
__global__ void predict_kernel(
		const int                 nbody,
		const Gravity::GParticle *ptcl,
		Gravity::GPredictor      *pred,
		const double              tsys)
{
	const int tid = threadIdx.x;
	const int off = blockDim.x * blockIdx.x;

	__shared__ Gravity::GParticle pshare[32];
	Gravity::GPredictor *prbuf = (Gravity::GPredictor *)pshare;

	{
		const double2 *src = (const double2 *)(ptcl+off);
		double2 *dst = (double2 *)(pshare);
		// copy 320 DP words
#pragma unrll
		for(int i=0; i<10; i++){
			dst[32*i + tid] = src[32*i + tid];
		}
	}
	Gravity::GPredictor pr;
	predict_one(tsys, pshare[tid], pr);
	prbuf[tid] = pr;
	{
		const double *src = (const double *)(prbuf);
		double *dst = (double *)(pred + off);
		// copy 160 DP words
#pragma unrll
		for(int i=0; i<10; i++){
			dst[32*i + tid] = src[32*i + tid];
		}
	}
}
#endif


void Gravity::predict_all(const double tsys){
	ptcl.htod(njpsend);
	// printf("sent %d stars\n", njpsend);

	const int ntpred = 32;
	
	const int nblock = (nbody/ntpred) + 
	                  ((nbody%ntpred) ? 1 : 0);
	predict_kernel <<<nblock, ntpred>>>
		(nbody, ptcl, pred, tsys);

	// pred.dtoh(); // THIS DEBUGGING LINE WAS THE BOTTLENECK
	// puts("pred all done");
	hipDeviceSynchronize(); // for profiling
}
enum{
	NJBLOCK = Gravity::NJBLOCK,
};

__device__ __forceinline__ void pp_interact(
		const Gravity::GPredictor &ipred,
		const Gravity::GPredictor &jpred,
		const double                eps2,
		double3                    &acc,
		double3                    &jrk,
		double3                    &snp)
{
		const double dx  = jpred.pos.x - ipred.pos.x;
		const double dy  = jpred.pos.y - ipred.pos.y;
		const double dz  = jpred.pos.z - ipred.pos.z;

		const double dvx = jpred.vel.x - ipred.vel.x;
		const double dvy = jpred.vel.y - ipred.vel.y;
		const double dvz = jpred.vel.z - ipred.vel.z;

		const double dax = jpred.acc.x - ipred.acc.x;
		const double day = jpred.acc.y - ipred.acc.y;
		const double daz = jpred.acc.z - ipred.acc.z;

		const double mj  = jpred.mass;

		const double dr2  = eps2 + dx*dx + dy*dy + dz*dz;
		const double drdv =  dx*dvx +  dy*dvy +  dz*dvz;
		const double dvdv = dvx*dvx + dvy*dvy + dvz*dvz;
		const double drda =  dx*dax +  dy*day +  dz*daz;

		const double rinv1 = rsqrt(dr2);
		const double rinv2 = rinv1 * rinv1;
		const double mrinv3 = mj * rinv1 * rinv2;

		double alpha = drdv * rinv2;
		double beta  = (dvdv + drda) * rinv2 + alpha * alpha;

		acc.x += mrinv3 * dx;
		acc.y += mrinv3 * dy;
		acc.z += mrinv3 * dz;

		alpha *= -3.0;
		const double  tx = dvx + alpha * dx;
		const double  ty = dvy + alpha * dy;
		const double  tz = dvz + alpha * dz;
		jrk.x += mrinv3 * tx;
		jrk.y += mrinv3 * ty;
		jrk.z += mrinv3 * tz;

		alpha *= 2.0;
		beta *= -3.0;
		snp.x += mrinv3 * (dax + alpha * tx + beta * dx);
		snp.y += mrinv3 * (day + alpha * ty + beta * dy);
		snp.z += mrinv3 * (daz + alpha * tz + beta * dz);
}

#if 0  // first version
__global__ void force_kernel(
		const int                  is,
		const int                  ie,
		const int                  nj,
		const Gravity::GPredictor *pred,
		const double               eps2,
		Gravity::GForce          (*fo)[NJBLOCK])
{
	const int xid = threadIdx.x + blockDim.x * blockIdx.x;
	const int yid = blockIdx.y;

	const int js = ((0 + yid) * nj) / NJBLOCK;
	const int je = ((1 + yid) * nj) / NJBLOCK;

	const int i = is + xid;
	if(i < ie){
		const Gravity::GPredictor ipred = pred[i];
		double3 acc = make_double3(0.0, 0.0, 0.0);
		double3 jrk = make_double3(0.0, 0.0, 0.0);
		double3 snp = make_double3(0.0, 0.0, 0.0);

#pragma unroll 4
		for(int j=js; j<je; j++){
			const Gravity::GPredictor &jpred = pred[j];
			pp_interact(ipred, jpred, eps2, acc, jrk, snp);
			
		}

		fo[xid][yid].acc = acc;
		fo[xid][yid].jrk = jrk;
		fo[xid][yid].snp = snp;
	}
}
#else // tuned with shared memory
__global__ void force_kernel(
		const int                  is,
		const int                  ie,
		const int                  nj,
		const Gravity::GPredictor *pred,
		const double               eps2,
		Gravity::GForce          (*fo)[NJBLOCK])
{
	const int tid = threadIdx.x;
	const int xid = threadIdx.x + blockDim.x * blockIdx.x;
	const int yid = blockIdx.y;

	const int js = ((0 + yid) * nj) / NJBLOCK;
	const int je = ((1 + yid) * nj) / NJBLOCK;
	const int je8 = js + 8*((je-js)/8);

	const int i = is + xid;

	__shared__ Gravity::GPredictor jpsh[8];

	const Gravity::GPredictor ipred = pred[i];
	double3 acc = make_double3(0.0, 0.0, 0.0);
	double3 jrk = make_double3(0.0, 0.0, 0.0);
	double3 snp = make_double3(0.0, 0.0, 0.0);

	for(int j=js; j<je8; j+=8){
		const double2 *src = (const double2 *)(pred + j);
		double2       *dst = (double2 *      )(jpsh);
		__syncthreads();
		if(tid < 40 /*sizeof(jpsh)/sizeof(double2)*/){
			dst[tid] = src[tid];
		}
		__syncthreads();
#pragma unroll
		for(int jj=0; jj<8; jj++){
			// const Gravity::GPredictor &jpred = pred[j+jj];
			const Gravity::GPredictor &jpred = jpsh[jj];
			pp_interact(ipred, jpred, eps2, acc, jrk, snp);
		}
	}
	const double2 *src = (const double2 *)(pred + je8);
	double2       *dst = (double2 *      )(jpsh);
	__syncthreads();
	if(tid < 40 /*sizeof(jpsh)/sizeof(double2)*/){
		dst[tid] = src[tid];
	}
	__syncthreads();
	for(int j=je8; j<je; j++){
		// const Gravity::GPredictor &jpred = pred[j];
		const Gravity::GPredictor &jpred = jpsh[j - je8];
		pp_interact(ipred, jpred, eps2, acc, jrk, snp);
	}

	if(i < ie){
		fo[xid][yid].acc = acc;
		fo[xid][yid].jrk = jrk;
		fo[xid][yid].snp = snp;
	}
}
#endif

__device__ double shfl_xor(const double x, const int bit){
	const int hi = __shfl_xor(__double2hiint(x), bit);
	const int lo = __shfl_xor(__double2loint(x), bit);
	return __hiloint2double(hi, lo);
}

__device__ double warp_reduce_double(double x){
	x += shfl_xor(x, 16);
	x += shfl_xor(x,  8);
	x += shfl_xor(x,  4);
	x += shfl_xor(x,  2);
	x += shfl_xor(x,  1);
	return x;
}

__global__ void reduce_kernel(
		const Gravity::GForce (*fpart)[NJBLOCK],
		Gravity::GForce        *ftot)
{
	const int bid = blockIdx.x;  // for particle
	const int xid = threadIdx.x; // for 30 partial force
	const int yid = threadIdx.y; // for 6 elements of Force

	const Gravity::GForce &fsrc = fpart[bid][xid];
	const double          *dsrc = (const double *)(&fsrc);
	
	const double x = xid<NJBLOCK ? dsrc[yid] : 0.0;
	const double y = warp_reduce_double(x);

	Gravity::GForce &fdst = ftot[bid];
	double          *ddst = (double *)(&fdst);
	if(32 == Gravity::NJREDUCE){
		if(0==xid) ddst[yid] = y;
	}
	if(64 == Gravity::NJREDUCE){
		// neeeds inter-warp reduction
		__shared__ double fsh[9][2];
		fsh[yid][xid/32] = y;
		__syncthreads();
		if(0==xid) ddst[yid] = fsh[yid][0] + fsh[yid][1];
	}
}

void Gravity::calc_force_in_range(
	   	const int    is,
		const int    ie,
		const double eps2,
		Force        force[] )
{
	assert(80 == sizeof(GPredictor));
	const int ni = ie - is;
	{
		const int niblock = (ni/NTHREAD) + 
						   ((ni%NTHREAD) ? 1 : 0);
		dim3 grid(niblock, NJBLOCK, 1);
		force_kernel <<<grid, NTHREAD>>>
			(is, ie, nbody, pred, eps2, fpart);
	}

	{
		// const int nwarp = 32;
		const int nword = sizeof(GForce) / sizeof(double);
		assert(9 == nword);
		reduce_kernel <<<ni, dim3(NJREDUCE, nword, 1)>>>
			(fpart, ftot);
	}

	ftot.dtoh(ni);
	for(int i=0; i<ni; i++){
		force[is+i].acc.x = ftot[i].acc.x;
		force[is+i].acc.y = ftot[i].acc.y;
		force[is+i].acc.z = ftot[i].acc.z;
		force[is+i].jrk.x = ftot[i].jrk.x;
		force[is+i].jrk.y = ftot[i].jrk.y;
		force[is+i].jrk.z = ftot[i].jrk.z;
		force[is+i].snp.x = ftot[i].snp.x;
		force[is+i].snp.y = ftot[i].snp.y;
		force[is+i].snp.z = ftot[i].snp.z;
	}
}

#include "pot-titan.hu"
