#include "hip/hip_runtime.h"
#include <cstdio>
#include "vector3.h"
#define CUDA_TITAN
#include "hermite8.h"
// #include "hermite8-titan.h"
#include "cuda-common.hu"

__device__ __forceinline__ void predict_one(
		const double             tsys,
		const Gravity::GParticle &p,
		Gravity::GPredictor      &pr)
{
		const double dt  = tsys - p.tlast;
		const double dt2 = (1./2.) * dt;
		const double dt3 = (1./3.) * dt;
		const double dt4 = (1./4.) * dt;
		const double dt5 = (1./5.) * dt;
		const double dt6 = (1./6.) * dt;
		const double dt7 = (1./7.) * dt;

		double3 pos, vel, acc, jrk;

		pos.x = 
			p.pos.x + dt *(
			p.vel.x + dt2*(
			p.acc.x + dt3*(
			p.jrk.x + dt4*(
			p.snp.x + dt5*(
			p.crk.x + dt6*(
			p.d4a.x + dt7*(
			p.d5a.x )))))));
		pos.y = 
			p.pos.y + dt *(
			p.vel.y + dt2*(
			p.acc.y + dt3*(
			p.jrk.y + dt4*(
			p.snp.y + dt5*(
			p.crk.y + dt6*(
			p.d4a.y + dt7*(
			p.d5a.y )))))));
		pos.z = 
			p.pos.z + dt *(
			p.vel.z + dt2*(
			p.acc.z + dt3*(
			p.jrk.z + dt4*(
			p.snp.z + dt5*(
			p.crk.z + dt6*(
			p.d4a.z + dt7*(
			p.d5a.z )))))));

		vel.x = 
			p.vel.x + dt *(
			p.acc.x + dt2*(
			p.jrk.x + dt3*(
			p.snp.x + dt4*(
			p.crk.x + dt5*(
			p.d4a.x + dt6*(
			p.d5a.x ))))));
		vel.y = 
			p.vel.y + dt *(
			p.acc.y + dt2*(
			p.jrk.y + dt3*(
			p.snp.y + dt4*(
			p.crk.y + dt5*(
			p.d4a.y + dt6*(
			p.d5a.y ))))));
		vel.z = 
			p.vel.z + dt *(
			p.acc.z + dt2*(
			p.jrk.z + dt3*(
			p.snp.z + dt4*(
			p.crk.z + dt5*(
			p.d4a.z + dt6*(
			p.d5a.z ))))));

		acc.x = 
			p.acc.x + dt *(
			p.jrk.x + dt2*(
			p.snp.x + dt3*(
			p.crk.x + dt4*(
			p.d4a.x + dt5*(
			p.d5a.x )))));
		acc.y = 
			p.acc.y + dt *(
			p.jrk.y + dt2*(
			p.snp.y + dt3*(
			p.crk.y + dt4*(
			p.d4a.y + dt5*(
			p.d5a.y )))));
		acc.z = 
			p.acc.z + dt *(
			p.jrk.z + dt2*(
			p.snp.z + dt3*(
			p.crk.z + dt4*(
			p.d4a.z + dt5*(
			p.d5a.z )))));

		jrk.x = 
			p.jrk.x + dt *(
			p.snp.x + dt2*(
			p.crk.x + dt3*(
			p.d4a.x + dt4*(
			p.d5a.x ))));
		jrk.y = 
			p.jrk.y + dt *(
			p.snp.y + dt2*(
			p.crk.y + dt3*(
			p.d4a.y + dt4*(
			p.d5a.y ))));
		jrk.z = 
			p.jrk.z + dt *(
			p.snp.z + dt2*(
			p.crk.z + dt3*(
			p.d4a.z + dt4*(
			p.d5a.z ))));

		pr.pos  = pos;
		pr.mass = p.mass;
		pr.vel  = vel;
		pr.acc  = acc;
		pr.jrk  = jrk;
}

#if 0 // naive version
__global__ void predict_kernel(
		const int                 nbody,
		const Gravity::GParticle *ptcl,
		Gravity::GPredictor      *pred,
		const double              tsys)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < nbody){
		Gravity::GParticle   p  = ptcl[tid];
		Gravity::GPredictor &pr = pred[tid];
		predict_one(tsys, p, pr);

	}
}
#else // specialized for 32 threads
// 26N DP -> 13N DP
__global__ void predict_kernel(
		const int                 nbody,
		const Gravity::GParticle *ptcl,
		Gravity::GPredictor      *pred,
		const double              tsys)
{
	const int tid = threadIdx.x;
	const int off = blockDim.x * blockIdx.x;

	__shared__ Gravity::GParticle pshare[32];
	Gravity::GPredictor *prbuf = (Gravity::GPredictor *)pshare;

	static_memcpy<double2, 32*13, 32> (pshare, ptcl+off);

	Gravity::GPredictor pr;
	predict_one(tsys, pshare[tid], pr);
	prbuf[tid] = pr;

	static_memcpy<double, 32*13, 32> (pred+off, prbuf);
}
#endif

void Gravity::predict_all(const double tsys){
	ptcl.htod(njpsend);
	// printf("sent %d stars\n", njpsend);

	const int ntpred = 32;
	
	const int nblock = (nbody/ntpred) + 
	                  ((nbody%ntpred) ? 1 : 0);
	predict_kernel <<<nblock, ntpred>>>
		(nbody, ptcl, pred, tsys);

	// pred.dtoh(); // THIS DEBUGGING LINE WAS THE BOTTLENECK
	// puts("pred all done");
	hipDeviceSynchronize(); // for profiling
}

enum{
	NJBLOCK = Gravity::NJBLOCK,
};

__device__ __forceinline__ void pp_interact(
		const Gravity::GPredictor &ipred,
		const Gravity::GPredictor &jpred,
		const double                eps2,
		double3                    &acc,
		double3                    &jrk,
		double3                    &snp,
		double3                    &crk)
{
		const double dx  = jpred.pos.x - ipred.pos.x;
		const double dy  = jpred.pos.y - ipred.pos.y;
		const double dz  = jpred.pos.z - ipred.pos.z;

		const double dvx = jpred.vel.x - ipred.vel.x;
		const double dvy = jpred.vel.y - ipred.vel.y;
		const double dvz = jpred.vel.z - ipred.vel.z;

		const double dax = jpred.acc.x - ipred.acc.x;
		const double day = jpred.acc.y - ipred.acc.y;
		const double daz = jpred.acc.z - ipred.acc.z;

		const double djx = jpred.jrk.x - ipred.jrk.x;
		const double djy = jpred.jrk.y - ipred.jrk.y;
		const double djz = jpred.jrk.z - ipred.jrk.z;

		const double mj  = jpred.mass;

		const double dr2  = eps2 + dx*dx + dy*dy + dz*dz;
		const double drdv =  dx*dvx +  dy*dvy +  dz*dvz;
		const double dvdv = dvx*dvx + dvy*dvy + dvz*dvz;
		const double drda =  dx*dax +  dy*day +  dz*daz;
		const double dvda = dvx*dax + dvy*day + dvz*daz;
		const double drdj =  dx*djx +  dy*djy +  dz*djz;

		const double rinv1 = rsqrt(dr2);
		const double rinv2 = rinv1 * rinv1;
		const double mrinv3 = mj * rinv1 * rinv2;

		double alpha = drdv * rinv2;
		double beta  = (dvdv + drda) * rinv2 + alpha * alpha;
		double gamma = (3.0*dvda + drdj)*rinv2 + alpha*(3.0*beta - 4.0*alpha*alpha);

		acc.x += mrinv3 * dx;
		acc.y += mrinv3 * dy;
		acc.z += mrinv3 * dz;

		alpha *= -3.0;
		const double  tx = dvx + alpha * dx;
		const double  ty = dvy + alpha * dy;
		const double  tz = dvz + alpha * dz;
		jrk.x += mrinv3 * tx;
		jrk.y += mrinv3 * ty;
		jrk.z += mrinv3 * tz;

		alpha *= 2.0;
		beta *= -3.0;
		const double ux = dax + alpha * tx + beta * dx;
		const double uy = day + alpha * ty + beta * dy;
		const double uz = daz + alpha * tz + beta * dz;
		snp.x += mrinv3 * ux;
		snp.y += mrinv3 * uy;
		snp.z += mrinv3 * uz;

		alpha *=1.5;
		beta  *= 3.0;
		gamma *= -3.0;
		crk.x += mrinv3 * (djx + alpha * ux + beta * tx + gamma * dx);
		crk.y += mrinv3 * (djy + alpha * uy + beta * ty + gamma * dy);
		crk.z += mrinv3 * (djz + alpha * uz + beta * tz + gamma * dz);
}

#if 0  // first version
__global__ void force_kernel(
		const int                  is,
		const int                  ie,
		const int                  nj,
		const Gravity::GPredictor *pred,
		const double               eps2,
		Gravity::GForce          (*fo)[NJBLOCK])
{
	const int xid = threadIdx.x + blockDim.x * blockIdx.x;
	const int yid = blockIdx.y;

	const int js = ((0 + yid) * nj) / NJBLOCK;
	const int je = ((1 + yid) * nj) / NJBLOCK;

	const int i = is + xid;
	if(i < ie){
		const Gravity::GPredictor ipred = pred[i];
		double3 acc = make_double3(0.0, 0.0, 0.0);
		double3 jrk = make_double3(0.0, 0.0, 0.0);
		double3 snp = make_double3(0.0, 0.0, 0.0);
		double3 crk = make_double3(0.0, 0.0, 0.0);

#pragma unroll 4
		for(int j=js; j<je; j++){
			const Gravity::GPredictor &jpred = pred[j];
			pp_interact(ipred, jpred, eps2, acc, jrk, snp, crk);
			
		}

		fo[xid][yid].acc = acc;
		fo[xid][yid].jrk = jrk;
		fo[xid][yid].snp = snp;
		fo[xid][yid].crk = crk;
	}
}
#else // tuned with shared memory
__global__ void force_kernel(
		const int                  is,
		const int                  ie,
		const int                  nj,
		const Gravity::GPredictor *pred,
		const double               eps2,
		Gravity::GForce          (*fo)[NJBLOCK])
{
	// const int tid = threadIdx.x;
	const int xid = threadIdx.x + blockDim.x * blockIdx.x;
	const int yid = blockIdx.y;

	const int js = ((0 + yid) * nj) / NJBLOCK;
	const int je = ((1 + yid) * nj) / NJBLOCK;
	const int je8 = js + 8*((je-js)/8);

	const int i = is + xid;

#if 1
	__shared__ Gravity::GPredictor jpsh[8];
#else
	__shared__ double2 align_buf[52];
	Gravity::GPredictor *jpsh = (Gravity::GPredictor *)(align_buf);
#endif

	const Gravity::GPredictor ipred = pred[i];
	double3 acc = make_double3(0.0, 0.0, 0.0);
	double3 jrk = make_double3(0.0, 0.0, 0.0);
	double3 snp = make_double3(0.0, 0.0, 0.0);
	double3 crk = make_double3(0.0, 0.0, 0.0);

	for(int j=js; j<je8; j+=8){
		__syncthreads();
		// static_memcpy<double2, 52, Gravity::NTHREAD> (jpsh, pred + j); // this doesn't run
		static_memcpy<double, 104, Gravity::NTHREAD> (jpsh, pred + j);
		// 104 = sizeof(jpsh) / sizeof(double)
		__syncthreads();

#pragma unroll
		for(int jj=0; jj<8; jj++){
			const Gravity::GPredictor &jpred = jpsh[jj];
			pp_interact(ipred, jpred, eps2, acc, jrk, snp, crk);
		}
	}

	__syncthreads();
	static_memcpy<double, 104, Gravity::NTHREAD> (jpsh, pred + je8);
	__syncthreads();

	for(int j=je8; j<je; j++){
		const Gravity::GPredictor &jpred = jpsh[j - je8];
		pp_interact(ipred, jpred, eps2, acc, jrk, snp, crk);
	}

	if(i < ie){
		fo[xid][yid].acc = acc;
		fo[xid][yid].jrk = jrk;
		fo[xid][yid].snp = snp;
		fo[xid][yid].crk = crk;
	}
}
#endif

__global__ void reduce_kernel(
		const Gravity::GForce (*fpart)[NJBLOCK],
		Gravity::GForce        *ftot)
{
	const int bid = blockIdx.x;  // for particle
	const int xid = threadIdx.x; // for 56 partial force
	const int yid = threadIdx.y; // for 12 elements of Force

	const Gravity::GForce &fsrc = fpart[bid][xid];
	const double          *dsrc = (const double *)(&fsrc);
	
	const double x = xid<NJBLOCK ? dsrc[yid] : 0.0;
	const double y = warp_reduce_double(x);

	Gravity::GForce &fdst = ftot[bid];
	double          *ddst = (double *)(&fdst);
	if(32 == Gravity::NJREDUCE){
		if(0==xid) ddst[yid] = y;
	}
	if(64 == Gravity::NJREDUCE){
		// neeeds inter-warp reduction
		__shared__ double fsh[12][2];
		fsh[yid][xid/32] = y;
		__syncthreads();
		if(0==xid) ddst[yid] = fsh[yid][0] + fsh[yid][1];
	}
}

void Gravity::calc_force_in_range(
	   	const int    is,
		const int    ie,
		const double eps2,
		Force        force[] )
{
	assert(104 == sizeof(GPredictor));
	const int ni = ie - is;
	{
		const int niblock = (ni/NTHREAD) + 
						   ((ni%NTHREAD) ? 1 : 0);
		dim3 grid(niblock, NJBLOCK, 1);
		force_kernel <<<grid, NTHREAD>>>
			(is, ie, nbody, pred, eps2, fpart);
	}

	{
		// const int nwarp = 32;
		const int nword = sizeof(GForce) / sizeof(double);
		assert(12 == nword);
		reduce_kernel <<<ni, dim3(NJREDUCE, nword, 1)>>>
			(fpart, ftot);
	}

	ftot.dtoh(ni);
	for(int i=0; i<ni; i++){
		force[is+i].acc.x = ftot[i].acc.x;
		force[is+i].acc.y = ftot[i].acc.y;
		force[is+i].acc.z = ftot[i].acc.z;
		force[is+i].jrk.x = ftot[i].jrk.x;
		force[is+i].jrk.y = ftot[i].jrk.y;
		force[is+i].jrk.z = ftot[i].jrk.z;
		force[is+i].snp.x = ftot[i].snp.x;
		force[is+i].snp.y = ftot[i].snp.y;
		force[is+i].snp.z = ftot[i].snp.z;
		force[is+i].crk.x = ftot[i].crk.x;
		force[is+i].crk.y = ftot[i].crk.y;
		force[is+i].crk.z = ftot[i].crk.z;
	}
}

#include "pot-titan.hu"

