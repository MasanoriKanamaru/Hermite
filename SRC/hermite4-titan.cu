#include "hip/hip_runtime.h"
#include <cstdio>
#include "vector3.h"
#define CUDA_TITAN
#include "hermite4.h"
// #include "hermite4-titan.h"

__global__ void predict_kernel(
		const int                 nbody,
		const Gravity::GParticle *ptcl,
		Gravity::GPredictor      *pred,
		const double              tsys)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < nbody){
		Gravity::GParticle   p  = ptcl[tid];
		Gravity::GPredictor &pr = pred[tid];

		const double dt = tsys - p.tlast;
		const double dt2 = (1./2.) * dt;;
		const double dt3 = (1./3.) * dt;;

		double3 pos, vel;
		pos.x = 
			p.pos.x + dt *(
			p.vel.x + dt2*(
			p.acc.x + dt3*(
			p.jrk.x )));
		pos.y = 
			p.pos.y + dt *(
			p.vel.y + dt2*(
			p.acc.y + dt3*(
			p.jrk.y )));
		pos.z = 
			p.pos.z + dt *(
			p.vel.z + dt2*(
			p.acc.z + dt3*(
			p.jrk.z )));
		vel.x = 
			p.vel.x + dt *(
			p.acc.x + dt2*(
			p.jrk.x ));
		vel.y = 
			p.vel.y + dt *(
			p.acc.y + dt2*(
			p.jrk.y ));
		vel.z = 
			p.vel.z + dt *(
			p.acc.z + dt2*(
			p.jrk.z ));

		pr.pos  = pos;
		pr.mass = p.mass;
		pr.vel  = vel;
	}
}

void Gravity::predict_all(const double tsys){
	ptcl.htod(njpsend);
	
	const int nblock = (nbody/NTHREAD) + 
	                  ((nbody%NTHREAD) ? 1 : 0);
	predict_kernel <<<nblock, NTHREAD>>>
		(nbody, ptcl, pred, tsys);

	pred.dtoh();
	puts("pred all done");
}

enum{
	NJBLOCK = Gravity::NJBLOCK,
};

__global__ void force_kernel(
		const int                  is,
		const int                  ie,
		const int                  nj,
		const Gravity::GPredictor *pred,
		const double               eps2,
		Gravity::GForce          (*fo)[NJBLOCK])
{
}

__device__ double shfl_xor(const double x, const int bit){
	const int hi = __shfl_xor(__double2hiint(x), bit);
	const int lo = __shfl_xor(__double2loint(x), bit);
	return __hiloint2double(hi, lo);
}

__device__ double warp_reduce_double(double x){
	x += shfl_xor(x, 16);
	x += shfl_xor(x,  8);
	x += shfl_xor(x,  4);
	x += shfl_xor(x,  2);
	x += shfl_xor(x,  1);
	return x;
}

__global__ void reduce_kernel(
		const Gravity::GForce (*fpart)[NJBLOCK],
		Gravity::GForce        *ftot)
{
	const int bid = blockIdx.x;  // for particle
	const int xid = threadIdx.x; // for 30 partial force
	const int yid = threadIdx.y; // for 6 elements of Force

	const Gravity::GForce &fsrc = fpart[bid][xid];
	const double          *dsrc = (const double *)(&fsrc);
	
	const double x = xid<NJBLOCK ? dsrc[yid] : 0.0;
	const double y = warp_reduce_double(x);

	Gravity::GForce &fdst = ftot[bid];
	double          *ddst = (double *)(&fdst);
	if(0==xid) ddst[yid] = y;
}

void Gravity::calc_force_in_range(
	   	const int    is,
		const int    ie,
		const double eps2,
		Force        force[] )
{
	const int ni = ie - is;
	const int niblock = (ni/NTHREAD) + 
	                   ((ni%NTHREAD) ? 1 : 0);
	dim3 grid(niblock, NJBLOCK, 1);
	force_kernel <<<grid, NTHREAD>>>
		(is, ie, nbody, pred, eps2, fpart);

	{
		const int nwarp = 32;
		const int nword = sizeof(GForce) / sizeof(double);
		assert(6 == nword);
		reduce_kernel <<<ni, dim3(nwarp, nword, 1)>>>
			(fpart, ftot);
	}
}

#include "pot-titan.hu"
