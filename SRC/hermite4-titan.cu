#include "hip/hip_runtime.h"
#include <cstdio>
#include "vector3.h"
#define CUDA_TITAN
#include "hermite4.h"
// #include "hermite4-titan.h"

__global__ void predict_kernel(
		const int                 nbody,
		const Gravity::GParticle *ptcl,
		Gravity::GPredictor      *pred,
		const double              tsys)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < nbody){
		Gravity::GParticle   p  = ptcl[tid];
		Gravity::GPredictor &pr = pred[tid];

		const double dt = tsys - p.tlast;
		const double dt2 = (1./2.) * dt;;
		const double dt3 = (1./3.) * dt;;

		double3 pos, vel;
		pos.x = 
			p.pos.x + dt *(
			p.vel.x + dt2*(
			p.acc.x + dt3*(
			p.jrk.x )));
		pos.y = 
			p.pos.y + dt *(
			p.vel.y + dt2*(
			p.acc.y + dt3*(
			p.jrk.y )));
		pos.z = 
			p.pos.z + dt *(
			p.vel.z + dt2*(
			p.acc.z + dt3*(
			p.jrk.z )));
		vel.x = 
			p.vel.x + dt *(
			p.acc.x + dt2*(
			p.jrk.x ));
		vel.y = 
			p.vel.y + dt *(
			p.acc.y + dt2*(
			p.jrk.y ));
		vel.z = 
			p.vel.z + dt *(
			p.acc.z + dt2*(
			p.jrk.z ));

		pr.pos  = pos;
		pr.mass = p.mass;
		pr.vel  = vel;
	}
}

void Gravity::predict_all(const double tsys){
	ptcl.htod(njpsend);
	
	const int nblock = (nbody/NTHREAD) + 
	                  ((nbody%NTHREAD) ? 1 : 0);
	predict_kernel <<<nblock, NTHREAD>>>
		(nbody, ptcl, pred, tsys);

	pred.dtoh();
	puts("pred all done");
	exit(1);
}

enum{
	NJBLOCK = Gravity::NJBLOCK,
};

__global__ void force_kernel(
		const int                  is,
		const int                  ie,
		const int                  nj,
		const Gravity::GPredictor *pred,
		const double               eps2,
		Gravity::GForce          (*fo)[NJBLOCK])
{
}

__global__ void reduce_kernel(
		const Gravity::GForce (*fpart)[NJBLOCK],
		Gravity::GForce        *ftot)
{
}

void Gravity::calc_force_in_range(
	   	const int    is,
		const int    ie,
		const double eps2,
		Force        force[] )
{
	const int ni = ie - is;
	const int niblock = (ni/NTHREAD) + 
	                   ((ni%NTHREAD) ? 1 : 0);
	dim3 grid(niblock, NJBLOCK, 1);
	force_kernel <<<grid, NTHREAD>>>
		(is, ie, nbody, pred, eps2, fpart);

	reduce_kernel <<<ni, NJREDUCE>>>
		(fpart, ftot);
}

#include "pot-titan.hu"
